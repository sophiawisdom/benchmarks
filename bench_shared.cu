
#include <hip/hip_runtime.h>
#include <assert.h>

#include "hiprand/hiprand_kernel.h"
#include "hip/hip_fp16.h"
#include "hip/hip_bf16.h"

#include <cmath>

#include "stdio.h"

#include <bit>

// shared b/w bench_global and bench_shared
enum Operation {
    ADD,
    INC,
    MAX,
    XOR,
    OR,
    EXCH,
    MUL,
    MANUAL_ADD,
    ADD_NOCHANGE,
    ADD_WARPCOALESCED
};

enum Strategy {
    TIDX,
    RAND
};

__forceinline__ __device__ __host__ float4 operator+(float4 a, float4 b) {
    return make_float4(a.x+b.x, a.y+b.y, a.z*b.z, a.w*b.w);
}

__device__ __forceinline__ float atomicMaxFloat(float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

template<typename dtype>
__device__ __forceinline__ dtype curand_cast(int val) {
    if constexpr (std::is_same<dtype, __half2>::value) {
        return __floats2half2_rn((float)val, (float)val);
    } else {
        return (dtype)val;
    }
}



#define ATOMIC_OP_FLOAT(loc, val, operation)                                  \
    {                                                                         \
        unsigned int *int_loc = reinterpret_cast<unsigned int*>(loc);         \
        unsigned int old_int = __float_as_int(*loc);                          \
        unsigned int assumed_int;                                             \
        unsigned int new_int;                                                 \
        do {                                                                  \
            assumed_int = old_int;                                            \
            new_int = __float_as_int(operation(__int_as_float(assumed_int), val)); \
            old_int = atomicCAS(int_loc, assumed_int, new_int);               \
        } while (assumed_int != old_int);                                     \
        return __int_as_float(old_int);                                       \
    }

#define ATOMIC_OP_DOUBLE(loc, val, operation)                                 \
    {                                                                         \
        unsigned long long *long_loc = reinterpret_cast<unsigned long long*>(loc); \
        unsigned long long old_long = __double_as_longlong(*loc);             \
        unsigned long long assumed_long;                                      \
        unsigned long long new_long;                                          \
        do {                                                                  \
            assumed_long = old_long;                                          \
            new_long = __double_as_longlong(operation(__longlong_as_double(assumed_long), val)); \
            old_long = atomicCAS(long_loc, assumed_long, new_long);           \
        } while (assumed_long != old_long);                                   \
        return __longlong_as_double(old_long);                                \
    }

#define ATOMIC_OP_GENERAL(loc, val, operation)                                 \
if constexpr (std::is_same<dtype, float>::value) {\
    ATOMIC_OP_FLOAT(loc, val, operation)\
} else if constexpr (std::is_same<dtype, double>::value) {\
    ATOMIC_OP_DOUBLE(loc, val, operation)\
} else {                                                                      \
    dtype old = *loc;\
    dtype assumed;\
    do {\
        assumed = old;\
        old = atomicCAS(loc, assumed, operation(assumed, val));\
    } while (assumed != old);\
    return old;\
}

template<typename dtype>
__device__ __forceinline__ dtype sophiaAtomicMul(dtype *loc, dtype val) {
    ATOMIC_OP_GENERAL(loc, val, [] __device__ (dtype a, dtype b) { return a * b; });
}

template<typename dtype>
__device__ __forceinline__ dtype sophiaManualAtomicAdd(dtype *loc, dtype val) {
    ATOMIC_OP_GENERAL(loc, val, [] __device__ (dtype a, dtype b) { return a + b; });
}

template<typename dtype, int shmem_size>
__device__ __forceinline__ void warpcoalesced_add(dtype *out, dtype rand) {
    if constexpr(std::is_same<dtype, unsigned int>::value) {
        if constexpr (shmem_size == 1) {
            unsigned int result;
            asm("redux.sync.add.u32 %0, %1, 0xffffffff;" : "=r"(result) : "r"(rand));
            if ((threadIdx.x % 32) == 0) {
                atomicAdd(out, result);
            }
        } else if constexpr (shmem_size == 2) {
            // do threads that don't participate in the reduction get the result?
            unsigned int result;
            asm("redux.sync.add.u32 %0, %1, 0xaaaaaaaa;" : "=r"(result) : "r"(rand));
            unsigned int result2;
            asm("redux.sync.add.u32 %0, %1, 0x55555555;" : "=r"(result2) : "r"(rand));
            if ((threadIdx.x % 32) == 0) {
                atomicAdd(out, result);
            }
            if ((threadIdx.x % 32) == 1) {
                atomicAdd(out, result2);
            }
        } else {
            unsigned int sum = rand;
            if constexpr (shmem_size == 4) {
                sum += __shfl_xor_sync(0xffffffff, sum, 4);
                sum += __shfl_xor_sync(0xffffffff, sum, 8);
                sum += __shfl_xor_sync(0xffffffff, sum, 16);
            } else if constexpr (shmem_size == 8) {
                sum += __shfl_xor_sync(0xffffffff, sum, 8);
                sum += __shfl_xor_sync(0xffffffff, sum, 16);
            } else if constexpr (shmem_size == 16) {
                sum += __shfl_xor_sync(0xffffffff, sum, 16);
            }

            if ((threadIdx.x % 32) < shmem_size) {
                atomicAdd(out, sum);
            }
        }
    } else if constexpr (std::is_same<dtype, float>::value) {
        float sum = rand;
        if constexpr (shmem_size == 1) {
            sum += __shfl_xor_sync(0xffffffff, sum, 1);
            sum += __shfl_xor_sync(0xffffffff, sum, 2);
            sum += __shfl_xor_sync(0xffffffff, sum, 4);
            sum += __shfl_xor_sync(0xffffffff, sum, 8);
            sum += __shfl_xor_sync(0xffffffff, sum, 16);
        } else if constexpr (shmem_size == 2) {
            sum += __shfl_xor_sync(0xffffffff, sum, 2);
            sum += __shfl_xor_sync(0xffffffff, sum, 4);
            sum += __shfl_xor_sync(0xffffffff, sum, 8);
            sum += __shfl_xor_sync(0xffffffff, sum, 16);
        } else if constexpr (shmem_size == 4) {
            sum += __shfl_xor_sync(0xffffffff, sum, 4);
            sum += __shfl_xor_sync(0xffffffff, sum, 8);
            sum += __shfl_xor_sync(0xffffffff, sum, 16);
        } else if constexpr (shmem_size == 8) {
            sum += __shfl_xor_sync(0xffffffff, sum, 8);
            sum += __shfl_xor_sync(0xffffffff, sum, 16);
        } else if constexpr (shmem_size == 16) {
            sum += __shfl_xor_sync(0xffffffff, sum, 16);
        }

        if ((threadIdx.x % 32) < shmem_size) {
            atomicAdd(out, sum);
        }
    }
}


template<typename dtype, Operation op, Strategy strat, int const_shmem_size>
__global__ void bench(int *outs, int shmem_size) {
    __shared__ dtype data[1024];

    unsigned int seed = threadIdx.x;
    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);

    for (int i = 0; i < shmem_size; i++) {
        if constexpr (std::is_same<dtype, __half2>::value) { // half2 needs different initializer
            data[i] = __floats2half2_rn(0, 0);
        } else if constexpr (op == ADD_NOCHANGE) {
            data[i] = INFINITY; // improves speed significantly
        } else {
            data[i] = 0;
        }
    }
    int our_index = threadIdx.x % shmem_size;
    if constexpr (strat == RAND) {
        our_index = hiprand(&state) % shmem_size;
    }

    __syncthreads();

    unsigned int start;
    asm volatile("mov.u32 %0, %clock;" : "=r"(start));

    for (int i = 0; i < 512; i++) {
        int rand_val = (op == INC) ? 1 : 15; // curand(&state);
        dtype rand = curand_cast<dtype>(rand_val);
        if constexpr (op == ADD || op == INC || op == ADD_NOCHANGE) {
            atomicAdd(&data[our_index], rand);
        } else if constexpr (op == MAX && (std::is_same<dtype, unsigned int>::value || std::is_same<dtype, unsigned long long>::value)) {
            atomicMax(&data[our_index], rand);
        } else if constexpr (op == MAX && (std::is_same<dtype, float>::value)) {
            atomicMaxFloat(&data[our_index], rand);
        } else if constexpr (op == XOR && (std::is_same<dtype, unsigned int>::value || std::is_same<dtype, unsigned long long>::value)) {
            atomicXor(&data[our_index], rand);
        } else if constexpr ((op == OR) && (std::is_same<dtype, unsigned int>::value || std::is_same<dtype, unsigned long long>::value)) {
            atomicOr(&data[our_index], rand);
        } else if constexpr (op == EXCH && (std::is_same<dtype, unsigned int>::value || std::is_same<dtype, unsigned long long>::value)) {
            atomicExch(&data[our_index], rand);
        } else if constexpr (op == MUL) {
            sophiaAtomicMul<dtype>(&data[our_index], rand);
        } else if constexpr (op == MANUAL_ADD) {
            sophiaManualAtomicAdd<dtype>(&data[our_index], rand);
        } else if constexpr (op == ADD_WARPCOALESCED) {
            static_assert(strat == TIDX);
            warpcoalesced_add<dtype, const_shmem_size>(&data[our_index], rand);
        }
    }

    unsigned int end;
    asm volatile("mov.u32 %0, %clock;" : "=r"(end));
    unsigned long long diff = end - start;

    if (threadIdx.x % 32 == 0) {
        outs[(blockIdx.x * blockDim.x + threadIdx.x)/32] = (int)diff;
    }
}

#define WARPCOALESCED(size) if (op == ADD_WARPCOALESCED && dtype == 0 && strat == 0 && shmem_size == size) {\
        kernel = &bench<float, ADD_WARPCOALESCED, TIDX, size>;\
    } else if (op == ADD_WARPCOALESCED && dtype == 2 && strat == 0) {\
        kernel = &bench<unsigned int, ADD_WARPCOALESCED, TIDX, size>;\
    }\


int bench_shared(
    int *outs,
    int op_arg,
    int shmem_size,
    int threads,
    int dtype,
    int strat
) {
    Operation op = (Operation)op_arg;
    using kernel_ptr = void(*)(int*, int);
    kernel_ptr kernel = nullptr;

    #define ASSIGN_KERNEL(DTYPE, TYPE_ID, OP) if (dtype == TYPE_ID && op == OP) {\
        if (strat == 0) {\
            kernel = &bench<DTYPE, OP, TIDX, 0>;\
        } else if (strat == 1) {\
            kernel = &bench<DTYPE, OP, RAND, 0>;\
        }\
    }

    ASSIGN_KERNEL(unsigned int, 1, ADD);
    ASSIGN_KERNEL(float, 0, ADD);
    ASSIGN_KERNEL(double, 3, ADD);
    ASSIGN_KERNEL(unsigned long long, 4, ADD);
    ASSIGN_KERNEL(__half2, 2, ADD);

    ASSIGN_KERNEL(unsigned int, 1, INC);

    ASSIGN_KERNEL(unsigned int, 1, MAX);
    ASSIGN_KERNEL(float, 0, MAX);
    ASSIGN_KERNEL(unsigned long long, 4, MAX);

    ASSIGN_KERNEL(unsigned int, 1, XOR);
    ASSIGN_KERNEL(unsigned long long, 4, XOR);

    ASSIGN_KERNEL(unsigned int, 1, OR);
    ASSIGN_KERNEL(unsigned long long, 4, OR);

    ASSIGN_KERNEL(unsigned int, 1, EXCH);
    ASSIGN_KERNEL(unsigned long long, 4, EXCH);

    ASSIGN_KERNEL(unsigned int, 1, MUL);
    ASSIGN_KERNEL(float, 0, MUL);
    ASSIGN_KERNEL(double, 3, MUL);
    ASSIGN_KERNEL(unsigned long long, 4, MUL);

    ASSIGN_KERNEL(unsigned int, 1, MANUAL_ADD);
    ASSIGN_KERNEL(float, 0, MANUAL_ADD);
    ASSIGN_KERNEL(double, 3, MANUAL_ADD);
    ASSIGN_KERNEL(unsigned long long, 4, MANUAL_ADD);

    ASSIGN_KERNEL(float, 0, ADD_NOCHANGE);
    ASSIGN_KERNEL(double, 3, ADD_NOCHANGE);

    WARPCOALESCED(1);
    WARPCOALESCED(2);
    WARPCOALESCED(4);
    WARPCOALESCED(8);
    WARPCOALESCED(16);
    WARPCOALESCED(32);
    WARPCOALESCED(64);
    WARPCOALESCED(128);
    if (op == ADD_WARPCOALESCED &&
        (shmem_size != 1) && (shmem_size != 2) && (shmem_size != 4) &&
        (shmem_size != 8) && (shmem_size != 16) && (shmem_size != 32) &&
        (shmem_size != 64) && (shmem_size != 128)) {
        printf("Couldn't run warpcoalesced op because shmem_size not in (1,2,4,8,16,32,64,128).\n");
        return 1;
    }

    if (kernel) {
        kernel<<<128, threads>>>(outs, shmem_size);
        return 0;
    } else {
        printf("RUNNING NOTHING for op=%d dtype=%d\n", op, dtype);
        return 1;
    }
}